#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#define BLOCK_SIZE 16
#define MAX_DISTANCE (1<<30-1)

namespace { // 为什么要namespace?

// __global__ void kernel(int n, int k, int *graph) {
//     auto i = blockIdx.y * blockDim.y + threadIdx.y;
//     auto j = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n && j < n) {
//         graph[i * n + j] = min(graph[i * n + j], graph[i * n + k] + graph[k * n + j]);
//     }
// }

}

__global__ void blocked_fw_phase1(const int blockId, int n, int* const graph) {
    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE]; // 对角块的shared存储, 即使是右下角块, 空间也是开够了的
    const int idx = threadIdx.x; // 块内的列号 // TODO 我担心这样是不是太浪费寄存器了
    const int idy = threadIdx.y;

    const int v1 = BLOCK_SIZE * blockId + idy; // 整个graph矩阵的行号
    const int v2 = BLOCK_SIZE * blockId + idx;

    int newPath;

    const int cellId = v1 * n + v2; // 在graph中对应的对角块的索引
    cacheGraph[idy][idx] = (v1 < n && v2 < n)?graph[cellId]:MAX_DISTANCE; // 处理右下角的对角块太小的问题, 避免越界
    #pragma unroll
    for (int u = 0; u < BLOCK_SIZE; ++u) { // FW的外层循环
        __syncthreads(); // 下一次循环前得同步
        newPath = cacheGraph[idy][u] + cacheGraph[u][idx]; // newPath省掉能不能节省一点寄存器?

        if (newPath < cacheGraph[idy][idx]) {
            cacheGraph[idy][idx] = newPath;
        }
    }

    if (v1 < n && v2 < n) {
        graph[cellId] = cacheGraph[idy][idx]; // 写回graph
    }
}

__global__ void blocked_fw_phase2(const int blockId, const int n, int* const graph) {
    if (blockIdx.x == blockId) return; // x的范围是0到2G-1, 交叉就是对角块, 是blockId

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    int v1 = BLOCK_SIZE * blockId + idy; // v1存的是对角块对应的位置
    int v2 = BLOCK_SIZE * blockId + idx;

    __shared__ int cacheGraphBase[BLOCK_SIZE][BLOCK_SIZE];

    // Load base block for graph and predecessors
    int cellId = v1 * n + v2;

    cacheGraphBase[idy][idx] = (v1 < n && v2 < n)?graph[cellId]:MAX_DISTANCE;

    // blockIdx.y==0对应的是行块, blockIdx.y==1对应的是列块
    if (blockIdx.y == 0) {
        v2 = BLOCK_SIZE * blockIdx.x + idx; // 行块需要修改列号, 列块需要修改行号
    } else {
        v1 = BLOCK_SIZE * blockIdx.x + idy;
    }

    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE]; // 把负责的graph块拷贝到了cacheGraph

    int currentPath;

    cellId = v1 * n + v2;
    currentPath = (v1 < n && v2 < n)?graph[cellId]:MAX_DISTANCE;

    cacheGraph[idy][idx] = currentPath; 

    __syncthreads();

    int newPath;

    if (blockIdx.y == 0) { // 处理行块
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraphBase[idy][u] + cacheGraph[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }

            cacheGraph[idy][idx] = currentPath;
            __syncthreads();
        }
    } else {
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraph[idy][u] + cacheGraphBase[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }

            cacheGraph[idy][idx] = currentPath;
            __syncthreads();
        }
    }

    if (v1 < n && v2 < n) {
        graph[cellId] = currentPath;
    }
}

__global__ void blocked_fw_phase3(const int blockId, const int n, int* const graph) {
    if (blockIdx.x == blockId || blockIdx.y == blockId) return; // 前两个阶段的块

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = blockDim.y * blockIdx.y + idy; // 在graph矩阵中负责的位置
    const int v2 = blockDim.x * blockIdx.x + idx;

    __shared__ int cacheGraphBaseRow[BLOCK_SIZE][BLOCK_SIZE]; // 对应的行块
    __shared__ int cacheGraphBaseCol[BLOCK_SIZE][BLOCK_SIZE]; // 对应的列块

    int v1Row = BLOCK_SIZE * blockId + idy; // 此元素对应的行块元素的行号
    int v2Col = BLOCK_SIZE * blockId + idx;


    cacheGraphBaseRow[idy][idx] = (v1Row < n && v2 < n)?graph[v1Row * n + v2]:MAX_DISTANCE;
    cacheGraphBaseCol[idy][idx] = (v1  < n && v2Col < n)?graph[v1 * n + v2Col]:MAX_DISTANCE;

    // Synchronize to make sure the all value are loaded in virtual block
   __syncthreads();

   int cellId;
   int currentPath;
   int newPath;

   // 不需要同步, 因为用不上
   if (v1  < n && v2 < n) {
       cellId = v1 * n + v2;
       currentPath = graph[cellId];

        #pragma unroll
       for (int u = 0; u < BLOCK_SIZE; ++u) {
           newPath = cacheGraphBaseCol[idy][u] + cacheGraphBaseRow[u][idx];
           if (currentPath > newPath) {
               currentPath = newPath;
           }
       }
       graph[cellId] = currentPath;
    } 
}

void apsp(int n, /* device */ int *graph) { // graph是device内存上的
    int numBlock = (n - 1) / BLOCK_SIZE + 1;
    dim3 gridPhase1(1 ,1, 1);
    dim3 gridPhase2(numBlock, 2 , 1);
    dim3 gridPhase3(numBlock, numBlock, 1);
    dim3 dimBlockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

    for (int blockID = 0; blockID < numBlock; ++blockID) {
         blocked_fw_phase1<<<gridPhase1, dimBlockSize>>>(blockID, n, graph);
         blocked_fw_phase2<<<gridPhase2, dimBlockSize>>>(blockID, n, graph);
         blocked_fw_phase3<<<gridPhase3, dimBlockSize>>>(blockID, n, graph);
    }
}

