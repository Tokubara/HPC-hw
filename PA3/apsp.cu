#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#define BLOCK_SIZE 16
#define MAX_DISTANCE (1<<30-1)

namespace { // 为什么要namespace?

// __global__ void kernel(int n, int k, int *graph) {
//     auto i = blockIdx.y * blockDim.y + threadIdx.y;
//     auto j = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n && j < n) {
//         graph[i * n + j] = min(graph[i * n + j], graph[i * n + k] + graph[k * n + j]);
//     }
// }

}

__global__ void blocked_fw_phase1(const int block_id, int n, int* const graph) {
    __shared__ int cache_diag[BLOCK_SIZE][BLOCK_SIZE]; // 对角块的shared存储, 即使是右下角块, 空间也是开够了的
    const int idx = threadIdx.x; // 块内的列号 // TODO 我担心这样是不是太浪费寄存器了
    const int idy = threadIdx.y;

    const int global_row_id = BLOCK_SIZE * block_id + idy; // 整个graph矩阵的行号
    const int global_col_id = BLOCK_SIZE * block_id + idx;

    int new_len;

    const int global_id = global_row_id * n + global_col_id; // 在graph中对应的对角块的索引
    cache_diag[idy][idx] = (global_row_id < n && global_col_id < n)?graph[global_id]:MAX_DISTANCE; // 处理右下角的对角块太小的问题, 避免越界
    #pragma unroll
    for (int u = 0; u < BLOCK_SIZE; ++u) { // FW的外层循环
        __syncthreads(); // 下一次循环前得同步
        new_len = cache_diag[idy][u] + cache_diag[u][idx]; // new_len省掉能不能节省一点寄存器?

        if (new_len < cache_diag[idy][idx]) {
            cache_diag[idy][idx] = new_len;
        }
    }

    if (global_row_id < n && global_col_id < n) {
        graph[global_id] = cache_diag[idy][idx]; // 写回graph
    }
}

__global__ void blocked_fw_phase2(const int block_id, const int n, int* const graph) {
    if (blockIdx.x == block_id) return; // x的范围是0到2G-1, 交叉就是对角块, 是block_id

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    int global_row_id = BLOCK_SIZE * block_id + idy; // global_row_id存的是对角块对应的位置
    int global_col_id = BLOCK_SIZE * block_id + idx;

    __shared__ int cache_diag[BLOCK_SIZE][BLOCK_SIZE];
    // Load base block for graph and predecessors
    int global_id = global_row_id * n + global_col_id;
    cache_diag[idy][idx] = (global_row_id < n && global_col_id < n)?graph[global_id]:MAX_DISTANCE;

    // blockIdx.y==0对应的是行块, blockIdx.y==1对应的是列块
    if (blockIdx.y == 0) {
        global_col_id = BLOCK_SIZE * blockIdx.x + idx; // 行块需要修改列号, 列块需要修改行号
    } else {
        global_row_id = BLOCK_SIZE * blockIdx.x + idy;
    }
    global_id = global_row_id * n + global_col_id;

    __shared__ int cache_self[BLOCK_SIZE][BLOCK_SIZE]; // 把负责的graph块拷贝到了cache_self
    int cur_len = (global_row_id < n && global_col_id < n)?graph[global_id]:MAX_DISTANCE;
    cache_self[idy][idx] = cur_len;  // 载入共享内存

    __syncthreads();

    int new_len;

    if (blockIdx.y == 0) { // 处理行块
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            new_len = cache_diag[idy][u] + cache_self[u][idx];

            if (new_len < cur_len) {
                cur_len = new_len;
            }

            cache_self[idy][idx] = cur_len;
            __syncthreads();
        }
    } else {
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            new_len = cache_self[idy][u] + cache_diag[u][idx];

            if (new_len < cur_len) {
                cur_len = new_len;
            }

            cache_self[idy][idx] = cur_len;
            __syncthreads();
        }
    }

    if (global_row_id < n && global_col_id < n) {
        graph[global_id] = cur_len;
    }
}

__global__ void blocked_fw_phase3(const int block_id, const int n, int* const graph) {
    if (blockIdx.x == block_id || blockIdx.y == block_id) return; // 前两个阶段的块

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int global_row_id = blockDim.y * blockIdx.y + idy; // 在graph矩阵中负责的位置
    const int global_col_id = blockDim.x * blockIdx.x + idx;
    const int cache_row_id = BLOCK_SIZE * block_id + idy;
    const int cache_col_id = BLOCK_SIZE * block_id + idx;

    __shared__ int cache_row[BLOCK_SIZE][BLOCK_SIZE]; // 对应的行块
    __shared__ int cache_col[BLOCK_SIZE][BLOCK_SIZE]; // 对应的列块

    cache_row[idy][idx] = (cache_row_id < n && global_col_id < n)?graph[cache_row_id * n + global_col_id]:MAX_DISTANCE; // 写对应的行块
    cache_col[idy][idx] = (global_row_id  < n && cache_col_id < n)?graph[global_row_id * n + cache_col_id]:MAX_DISTANCE; // 写对应的列块

   __syncthreads();
   
   if (global_row_id  < n && global_col_id < n) {
       int global_id = global_row_id * n + global_col_id;
       int cur_len = graph[global_id];

       int new_len;

        #pragma unroll
       for (int u = 0; u < BLOCK_SIZE; ++u) {
           new_len = cache_col[idy][u] + cache_row[u][idx];
           if (cur_len > new_len) {
               cur_len = new_len;
           } // 不需要同步, 因为用不上
       }
       graph[global_id] = cur_len;
    } 
}

void apsp(int n, /* device */ int *graph) { // graph是device内存上的
    int num_block = (n - 1) / BLOCK_SIZE + 1;
    dim3 grid_phase1(1 ,1, 1);
    dim3 grid_phase2(num_block, 2 , 1);
    dim3 grid_phase3(num_block, num_block, 1);
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);

    for (int blockID = 0; blockID < num_block; ++blockID) {
         blocked_fw_phase1<<<grid_phase1, block_size>>>(blockID, n, graph);
         blocked_fw_phase2<<<grid_phase2, block_size>>>(blockID, n, graph);
         blocked_fw_phase3<<<grid_phase3, block_size>>>(blockID, n, graph);
    }
}

